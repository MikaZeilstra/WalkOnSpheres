#include "hip/hip_runtime.h"
﻿
#pragma once
#include "kernel.cuh"
#include "vec_mult.cuh"

ihipStream_t* stream;
hipEvent_t start;
hipEvent_t end;


//Sets values of image to initial values
__global__ void set_initial_distance_kernel(float4* image, uint2* size)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < size->x && y < size->y) {
        image[x + y*size->x] = { DISTANCE_INTIAL,DISTANCE_INTIAL,DISTANCE_INTIAL, DISTANCE_INTIAL};
    }


}

//Get point for parameter t on bezier curve with control points control_points[0] to control_points[3]
__device__ __inline__ float3 get_bezier_point(float3* control_points, float t) {
    return  (1 - t) * (1 - t) * (1 - t) * control_points[0] +
        3 * (1 - t) * (1 - t) * t * control_points[1] +
        3 * (1 - t) * t * t * control_points[2] +
        t * t * t * control_points[3];
}

//Get analythical normal for parameter t on bezier curve with control points control_points[0] to control_points[3]
__device__ __inline__ float3 get_bezier_normal(float3* control_points, float t) {
    float3 tangent =
        -3 * (1 - t) * (1 - t) * control_points[0] +
        3 * (1 - t) * (1 - t) * control_points[1] - 6 * t * (1 - t) * control_points[1] -
        3 * t * 2 * control_points[2] +
        6 * t * (1 - t) * control_points[2] +
        3 * t * 2 * control_points[3];

    //Rotate tangent (cc) left and return it
    return { tangent.y,-tangent.x };
}

// find the local minimum of distance of point the the bezier curve with control points control_points[0] to control_points[3] between parameter min_x and max_x
__device__ float find_local_minimum(float3* control_points,float3 point, float min_x, float max_x) {
    //Binary search for minimum
    //Set inital boundaries
    float m = min_x;
    float n = max_x;
    //Initialize middle
    float k = 0;
    //Continue untill close enough
    while(abs(n-m) > LOCAL_MINIMUM_EPS) {
        //Set k to middle of interval
        k = m + (n - m) / 2;
        //check if left or right sight of middle is closer and move other interval boundary to middle
        if (sqr_dist(get_bezier_point(control_points, k - LOCAL_MINIMUM_EPS),point) < sqr_dist(get_bezier_point(control_points, k + LOCAL_MINIMUM_EPS),point)) {
            n = k;
        } else {
            m = k;
        }
    }

    //Return the parameter value of the closest point on the bezier curve
    return k;
}

//Finds the parameter value of the closest point on a bezier curve with control points control_points[0] to control_points[3]  to point 
__device__ float find_closest_bezier_point(float3* control_points,float3 point) {
    //Setup temp vars
    float mindex = 0; //index of local minium in discretized domain
    float min_dist = 10e5; //Value of the closest distance
    float c_dist = 0; //Temp var for distance of current interval
    float t = 0; //Parameter of the curve we are looking at

    //Linearly scan for segment with closest point
    #pragma unroll
    //divide curve into NUM_LIN_MINIMUM_SCANS and find closest interval
    for (int i = 0; i <= NUM_LIN_MINIMUM_SCANS; i++) {
        //Set next value of paramter on curve
        t = i * (1.0f / NUM_LIN_MINIMUM_SCANS);
        //Get the distance of current paramter from point to curvepoint
        c_dist = sqr_dist(get_bezier_point(control_points, t), point);
        //if current distance is smaller than minimum update it
        if (c_dist < min_dist) {
            mindex = i;
            min_dist = c_dist;
        }
    }

    //Get local minimum inside interval using binary search and return in
    return find_local_minimum(control_points, point, fmaxf((mindex - 1) * (1.0f / NUM_LIN_MINIMUM_SCANS), 0), fminf((mindex + 1) * (1.0f / NUM_LIN_MINIMUM_SCANS), 1));
}

//interpolate the curve color to the given parameter t
__device__ float3 find_color(uint2& index, float t,float3* colors, float*color_t) {
    
    //Set index to start of current curve colors
    int i = index.x;
    //Scan for the colro value which is the largest to the parameter without going over
    while (i < index.x + index.y && color_t[i + 1] < t) {
        i++;
    }

    //find ratio for linear interpolation
    float r = (t - color_t[i]) / (color_t[i + 1] - color_t[i]);

    //Return interpolated value
    return {
        (colors[i].x * (1 - r)) + (colors[i + 1].x * r)  ,
        (colors[i].y * (1 - r)) + (colors[i + 1].y * r),
        (colors[i].z * (1 - r)) + (colors[i + 1].z * r)
    };
}

// Kernel for making the distance map 
__global__ void make_distance_map_kernel(float4* distance,float4* color, curve_info* curve_pointers)
{
    //Get x and y of this thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //convert x and y into the point we are finding the distance for
    float3 point = { (float)x / curve_pointers->image_size->x , (float)y / curve_pointers->image_size->y };

    if (x < curve_pointers->image_size->x && y < curve_pointers->image_size->y) {
        // For each curve segment calculate the distance to this point
        for (int segment_index = 0; segment_index < *(curve_pointers->number_of_segments); segment_index++) {

            //Calculate distance to the bounding box of the curve
            float dx = fmaxf(abs(curve_pointers->bounding_boxes[segment_index * 2].x - point.x) - curve_pointers->bounding_boxes[(segment_index * 2) + 1].x / 2, 0);
            float dy = fmaxf(abs(curve_pointers->bounding_boxes[segment_index * 2].y - point.y) - curve_pointers->bounding_boxes[(segment_index * 2) + 1].y / 2, 0);
            float bb_distance = sqrtf(dx * dx + dy * dy);

            //If the distance to the bounding box is smaller than the current minimum find the actual distance to the segment
            if (bb_distance < distance[x + y * curve_pointers->image_size->x].w) {
                float closest_t = find_closest_bezier_point(&(curve_pointers->control_points[segment_index * 4]), point);
                float3 curve_point = get_bezier_point(&(curve_pointers->control_points[segment_index * 4]), closest_t);
                float curve_distance = sqrtf(sqr_dist(curve_point, point));
                                
                //If the actual distance is also smaller update the distance and color map
                if (curve_distance < distance[x + y * curve_pointers->image_size->x].w) {
                    float3 curve_normal = get_bezier_normal(&(curve_pointers->control_points[segment_index * 4]), closest_t);
                    float3 curve_color = { 0,0,0 };
                    unsigned int curve_index = curve_pointers->curve_map[segment_index];
                    float curve_t = closest_t + curve_pointers->curve_index[segment_index];
                    // find the color with the correct side
                    if (dot_prod(curve_normal, curve_point - point) < 0) {
                        curve_color = find_color(curve_pointers->color_right_index[curve_index], curve_t, curve_pointers->color_right, curve_pointers->color_right_u);
                    }
                    else {
                        curve_color = find_color(curve_pointers->color_left_index[curve_index], curve_t, curve_pointers->color_left, curve_pointers->color_left_u);
                    }

                    //Set the color and distance maps to found values
                    color[x + y * curve_pointers->image_size->x] = { curve_color.x,curve_color.y,curve_color.z,fmaxf(curve_distance- DISTANCE_MAP_EPS,0)};
                    distance[x + y * curve_pointers->image_size->x] = { curve_distance,curve_distance,curve_distance,curve_distance };

                    //If we are close enough to a curve draw it in the boundaries frame
                    if (curve_distance < DISTANCE_MAP_EPS) {
                        curve_pointers->boundaries[x + y * curve_pointers->image_size->x] = { curve_color.x,curve_color.y,curve_color.z,1 };
                    }

                }
                
            }

        }
    }


}


//Linear bilear interpolation of distance map
__device__ float4 interpolate_bilinear(float4* distance_map,uint2* size, float3& point){
    // find x and y as non-normalized coordinates
    float x = point.x * size->x;
    float y = point.y * size->y;

    //Get known points around point
    int2 p1 = { (int)floorf(x),(int)floorf(y) };
    int2 p2 = { (int)ceilf(x),(int)floorf(y) };
    int2 p3 = { (int)floorf(x),(int)ceilf(y) };
    int2 p4 = { (int)ceilf(x),(int)ceilf(y) };

    //interpolate
    float x_r = point.x - p1.x;
    float y_r = point.y - p1.y;

    float4 v1 = (1 - x_r) * distance_map[p1.y + p1.x * size->y] + x_r * distance_map[p2.y + p2.x * size->y];
    float4 v2 = (1 - x_r) * distance_map[p3.y + p3.x * size->y] + x_r * distance_map[p4.y + p4.x * size->y];

    float4 v3 = (1 - y_r) * v1 + (y_r)*v2;

    //Set distance to lowest value to avoid going over curves due to erro
    //v3.w = fminf(fminf(distance_map[p1.x + p1.y * size->x].w, distance_map[p2.x + p2.y * size->x].w), fminf(distance_map[p3.x + p3.y * size->x].w, distance_map[p4.x + p4.y * size->x].w));

    //return value
    return v3;

}

//Kernel used for sampeling
__global__ void sample_kernel(curve_info* curve_pointers, unsigned int sample_count) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //normalize coordinates as staring point
    float3 point = { (float)x / curve_pointers->image_size->x , (float)y / curve_pointers->image_size->y };

    //Find cloest value 
    float4 value = curve_pointers->boundary_conditions[x + y * curve_pointers->image_size->x];
    //Init temp vars
    float rot_cos = 0;
    float rot_sin = 1;
    int i = 0;

    //only sample if x and y are within image
    if (x < curve_pointers->image_size->x && y < curve_pointers->image_size->y) {
        //Walk while not close enough and not reached max walks
        while(value.w > WALKING_SPHERES_EPS && i < WALKING_SPHERES_MAX_WALK) {
            //get random cos and sin
            sincospif(2*hiprand_uniform(&curve_pointers->rand_state[x+y* curve_pointers->image_size->x]), &rot_sin, &rot_cos);

            //Go to new point on circle with radius distance around point
            point = { fminf(fmaxf(point.x + rot_cos * value.w,0),1.0f), fminf(fmaxf(point.y + rot_sin * value.w,0),1.0f),0 };
            
            //Use neirest neighbour interpolation since linear interpolation was found to not be worth performance/quality trade-off
            value = curve_pointers->boundary_conditions[min(max((int)round(point.x * curve_pointers->image_size->x), 0), curve_pointers->image_size->x) +  min(max((int)round(point.y * curve_pointers->image_size->y), 0), (curve_pointers->image_size->y) - 1) * curve_pointers->image_size->x];
            i++;
        }
    }
    //Set final color
    float4 final_v = { value.x, value.y,value.z, 1};
    //Add it to the accumulator
    curve_pointers->sample_accumulator[x + y * curve_pointers->image_size->x] = curve_pointers->sample_accumulator[x + y * curve_pointers->image_size->x] + final_v;
    
    //Set current solution to accumulator divided by number of samples
    curve_pointers->current_solution[x + y * curve_pointers->image_size->x] = ((1.0f) / sample_count) * curve_pointers->sample_accumulator[x + y * curve_pointers->image_size->x];
    __syncthreads;
    //If the pixel is not on the edge calculate the laplacian
    if (x >= 1 && x < curve_pointers->image_size->x - 1 && y >= 1 && y < curve_pointers->image_size->y) {
        curve_pointers->laplacian_mag[x + y * curve_pointers->image_size->x] =
            4 * curve_pointers->current_solution[x + y * curve_pointers->image_size->x] -
            curve_pointers->current_solution[x + 1 + y * curve_pointers->image_size->x] -
            curve_pointers->current_solution[x - 1 + y * curve_pointers->image_size->x] -
            curve_pointers->current_solution[x + (y + 1) * curve_pointers->image_size->x] -
            curve_pointers->current_solution[x + (y - 1) * curve_pointers->image_size->x];
    }
    
}

//Kernel for setting up hiprand states
__global__ void setup_curand_kernel(hiprandState_t* states, uint2* size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < size->x && y < size->y) {
        hiprand_init(x + y * size->x, size->x, size->y, &states[x + y * size->x]);
    }
}

//Kernel for reseting accumulator
__global__ void reset_sample_kernel (curve_info* curve_pointers) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < curve_pointers->image_size->x && y < curve_pointers->image_size->y) {
        curve_pointers->sample_accumulator[x + y * curve_pointers->image_size->x] = { 0,0,0,1 };
    }
}

//kernel for drawing circles
__global__ void create_circle_kernel(curve_info*curve_pointers,float4* image, float2 circle_centre, float radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    //Find center of circle by normalizing thread coords
    float x_normal = x / (float)curve_pointers->image_size->x;
    float y_normal = y / (float)curve_pointers->image_size->y;

    //If inside the image draw the circle
    if (x < curve_pointers->image_size->x && y < curve_pointers->image_size->y) {

        //find distanc to circle center
        float circle_dist = sqrtf((circle_centre.x - x_normal) * (circle_centre.x - x_normal) + (circle_centre.y - y_normal) * (circle_centre.y - y_normal));

        //If the distanc is around the radius make mix some circle color into the image at this point
        if (circle_dist < radius + CIRCLE_WIDTH && circle_dist > radius - CIRCLE_WIDTH) {
            image[x + y * curve_pointers->image_size->x] = 0.5 * image[x + y * curve_pointers->image_size->x] + 0.5 * make_float4( CIRCLE_COLOR );
        }
        //Else if it is close to the circle center mix some center color into the image
        else if(circle_dist < CIRCLE_WIDTH*5) {
            image[x + y * curve_pointers->image_size->x] = 0.5 * image[x + y * curve_pointers->image_size->x] + 0.5 * make_float4(CIRCLE_CENTER_COLOR);
        }
    }
}

//Wrapper to call the kernels easily
namespace KernelWrapper{
    void set_initial_distance(uint2 size, float4* image_device, uint2* size_device) {
        //calculate the blocks needed per kernel
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int) ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int) ceil(size.y / THREADS_PER_BLOCK)) ,1};

        //Call kernel
        set_initial_distance_kernel << <dim_block_grid, dim_threads_per_block,0,stream >> > (image_device,size_device);
    }

    void make_distance_map(uint2 size, float4* distance_device,float4* color_device,  curve_info* curve_pointers) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int)ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int)ceil(size.y / THREADS_PER_BLOCK)) ,1 };

        make_distance_map_kernel << <dim_block_grid, dim_threads_per_block, 0, stream >> > (distance_device,color_device,  curve_pointers);
    }

    void sample(uint2 size, curve_info* curve_info_device, unsigned int sample_count) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int)ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int)ceil(size.y / THREADS_PER_BLOCK)) ,1 };

        sample_kernel << <dim_block_grid, dim_threads_per_block, 0, stream >> > (curve_info_device, sample_count);
    }

    void setup_curand(uint2 size, hiprandState_t* states, uint2* size_device) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int)ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int)ceil(size.y / THREADS_PER_BLOCK)) ,1 };

        setup_curand_kernel << <dim_block_grid, dim_threads_per_block, 0, stream >> > (states, size_device);
    }

    void reset_samples(uint2 size, curve_info* curve_info_device) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int)ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int)ceil(size.y / THREADS_PER_BLOCK)) ,1 };

        reset_sample_kernel << <dim_block_grid, dim_threads_per_block, 0, stream >> > (curve_info_device);
    }

    void create_circle(uint2 size, curve_info* curve_info_device, float4* image, float2 circle_center, float radius) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int)ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int)ceil(size.y / THREADS_PER_BLOCK)) ,1 };

        create_circle_kernel << <dim_block_grid, dim_threads_per_block, 0, stream >> > ( curve_info_device, image, circle_center,radius);
    }
}

//GPU utility functions
void GPU_setup() {
    hipFree(0);
    CALL_CHECK(hipStreamCreate(&stream));
    CALL_CHECK(hipEventCreate(&start)); 
    CALL_CHECK(hipEventCreate(&end));
}

void* GPU_malloc(size_t size) {
    void* device_ptr = nullptr;
    CALL_CHECK(hipMallocAsync(reinterpret_cast<void**>(&device_ptr), size, stream));
    return device_ptr;
}

void* GPU_upload(size_t size, void* data) {
    void* device_ptr = nullptr;
    CALL_CHECK(hipMallocAsync(&device_ptr, size, stream));
    CALL_CHECK(hipMemcpyAsync(
        device_ptr,
        data,
        size,
        hipMemcpyHostToDevice,
        stream
    ));
    return device_ptr;
}

void GPU_download(size_t size, void* data_device, void* dst) {
    CALL_CHECK(hipMemcpyAsync(
        dst,
        data_device,
        size,
        hipMemcpyDeviceToHost,
        stream
    ));
}


void GPU_free(void* device_ptr) {
    hipFreeAsync(device_ptr, stream);
}

void GPU_sync() {
    CALL_CHECK(hipDeviceSynchronize());
    CALL_CHECK(hipStreamSynchronize(stream));
}

void GPU_copy(rsize_t size, void* data_src, void* data_dst) {
    CALL_CHECK(hipMemcpyAsync(
        data_dst,
        data_src,
        size,
        hipMemcpyDeviceToDevice,
        stream
    ));
}

void GPU_start_timer() {
    CALL_CHECK(hipEventRecord(start, stream));
}

float GPU_stop_timer(std::string description, bool use_newline) {
    float time = 0;

    GPU_sync();
    CALL_CHECK(hipEventRecord(end, stream));
    GPU_sync();
    CALL_CHECK(hipEventElapsedTime(&time, start, end));

    std::cout << "Duration of " << description << " : " << time << " ms" << (use_newline ? "\n" : "") << std::flush;

    return time;
}