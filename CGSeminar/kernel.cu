#include "hip/hip_runtime.h"
﻿
#pragma once
#include "kernel.cuh"
#include "vec_mult.cuh"

#define THREADS_PER_BLOCK 32
#define DISTANCE_INTIAL 10e5
#define LOCAL_MINIMUM_EPS 1e-2
#define NUM_LIN_MINIMUM_SCANS  10
#define NUM_BIN_MINIMUM_SCANS 100

ihipStream_t* stream;

__global__ void set_initial_distance_kernel(float4* image, uint2* size)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < size->x && y < size->y) {
        image[x + y*size->y] = { DISTANCE_INTIAL,DISTANCE_INTIAL,DISTANCE_INTIAL, DISTANCE_INTIAL};
    }


}

__device__ __inline__ float3 get_bezier_point(float3* control_points, float t) {
    return  (1 - t) * (1 - t) * (1 - t) * control_points[0] +
        3 * (1 - t) * (1 - t) * t * control_points[1] +
        3 * (1 - t) * t * t * control_points[2] +
        t * t * t * control_points[3];
}

__device__ __inline__ float3 get_bezier_normal(float3* control_points, float t) {
    float3 tangent =
        -3 * (1 - t) * (1 - t) * control_points[0] +
        3 * (1 - t) * (1 - t) * control_points[1] - 6 * t * (1 - t) * control_points[1] -
        3 * t * 2 * control_points[2] +
        6 * t * (1 - t) * control_points[2] +
        3 * t * 2 * control_points[3];

    return { -tangent.y,tangent.x };
}

__device__ float find_local_minimum(float3* control_points,float3 point, float min_x, float max_x) {
    float m = min_x;
    float n = max_x;
    float k = 0;
    #pragma unroll
    while(abs(n-m) > LOCAL_MINIMUM_EPS) {
        k = m + (n - m) / 2;
        if (sqr_dist(get_bezier_point(control_points, k - LOCAL_MINIMUM_EPS),point) < sqr_dist(get_bezier_point(control_points, k + LOCAL_MINIMUM_EPS),point)) {
            n = k;
        } else {
            m = k;
        }
    }
    return k;
}

__device__ float find_closest_bezier_point(float3* control_points,float3 point) {
    float mindex = 0;
    float min_dist = 10e10;
    float c_dist = 0;
    float t = 0;
    #pragma unroll
    for (int i = 0; i <= NUM_LIN_MINIMUM_SCANS; i++) {
        t = i * (1.0f / NUM_LIN_MINIMUM_SCANS);
        c_dist = sqr_dist(get_bezier_point(control_points, t), point);
        if (c_dist < min_dist) {
            mindex = i;
            min_dist = c_dist;
        }
    }
    return find_local_minimum(control_points, point, fmaxf((mindex - 1) * (1.0f / NUM_LIN_MINIMUM_SCANS), 0), fminf((mindex + 1) * (1.0f / NUM_LIN_MINIMUM_SCANS), 1));
}


__global__ void make_distance_map_kernel(float4* image, uint2* size, curve_info* curve_pointers)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float3 point = { (float)x / size->x , (float)y / size->y };

    if (x < size->x && y < size->y) {
        // For each curve segment calculate the distance to this point
        for (int segment_index = 0; segment_index < *(curve_pointers->number_of_segments); segment_index++) {

            //Calculate distance to the bounding box of the curve
            float dx = fmaxf(abs(curve_pointers->bounding_boxes[segment_index * 2].x - point.x) - curve_pointers->bounding_boxes[(segment_index * 2) + 1].x / 2, 0);
            float dy = fmaxf(abs(curve_pointers->bounding_boxes[segment_index * 2].y - point.y) - curve_pointers->bounding_boxes[(segment_index * 2) + 1].y / 2, 0);
            float bb_distance = sqrtf(dx * dx + dy * dy);

            //If the distance to the bounding box is smaller than the current minimum find the actual distance to the segment
            if (bb_distance < image[x + y * size->y].x) {
                float closest_t = find_closest_bezier_point(&(curve_pointers->control_points[segment_index * 4]), point);
                float distance = sqrtf(sqr_dist(get_bezier_point(&(curve_pointers->control_points[segment_index * 4]), closest_t), point));
                
                //If the actual distance is also smaller update the distance and color map
                if (distance < image[x + y * size->y].x) {
                    image[x + y * size->y] = { distance,distance,distance,1 };
                }
                
            }

        }
    }


}




namespace KernelWrapper{
    void set_initial_distance(uint2 size, float4* image_device, uint2* size_device) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int) ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int) ceil(size.y / THREADS_PER_BLOCK)) ,1};

        set_initial_distance_kernel << <dim_block_grid, dim_threads_per_block,0,stream >> > (image_device,size_device);
    }

    void make_distance_map(uint2 size, float4* image_device, uint2* size_device, curve_info* curve_pointers) {
        dim3 dim_threads_per_block = { THREADS_PER_BLOCK ,THREADS_PER_BLOCK, 1 };
        dim3 dim_block_grid = { ((unsigned int)ceil(size.x / THREADS_PER_BLOCK)), ((unsigned int)ceil(size.y / THREADS_PER_BLOCK)) ,1 };

        make_distance_map_kernel << <dim_block_grid, dim_threads_per_block, 0, stream >> > (image_device, size_device, curve_pointers);
    }

}

void GPU_setup() {
    hipFree(0);

    CALL_CHECK(hipStreamCreate(&stream));
}

void* GPU_upload(size_t size, void* data) {
    void* device_ptr = nullptr;
    CALL_CHECK(hipMallocAsync(&device_ptr, size, stream));
    CALL_CHECK(hipMemcpyAsync(
        device_ptr,
        data,
        size,
        hipMemcpyHostToDevice,
        stream
    ));
    return device_ptr;
}

void GPU_download(size_t size, void* data_device, void* dst) {
    CALL_CHECK(hipMemcpyAsync(
        dst,
        data_device,
        size,
        hipMemcpyDeviceToHost,
        stream
    ));
}


void GPU_free(void* device_ptr) {
    hipFreeAsync(device_ptr, stream);
}

void GPU_sync() {
    CALL_CHECK(hipDeviceSynchronize());
    CALL_CHECK(hipStreamSynchronize(stream));
}